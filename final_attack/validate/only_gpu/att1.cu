#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <iterator>

#include "../../common.h"
#include "../../cipher.cu"
//#include "../../mygrain_lib.cuh"
#include "../../utils.cu"


unsigned int TRIES;
unsigned int INSTANCES;
unsigned int M_d;
unsigned int N;



__device__ uint8_t d_k[K_dim]={0x0};
__constant__ uint8_t d_p0_dev;
__constant__ uint8_t M_d_dev;
__constant__ uint8_t k_curr_dev;


typedef struct cubes {
	uint8_t *cubes;
	uint8_t *cubes_len;
	uint64_t n_cubes;
 } cubes;

 typedef struct superpolys {
	uint8_t *arr=NULL;
	uint8_t *arr_lens=NULL;
	uint64_t ncubes=0;
} superpolys;
/*
  Kernel for selecting different random k
*/

__global__ void random_k(uint8_t *out_k,hiprandState *states){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(clock64(), tid, 0, &states[tid]);
  out_k[tid] = hiprand_uniform(&states[tid])*(M_d_dev)+1;
}

__global__ void random_I(uint8_t *out_I,hiprandState *states){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(clock64(), tid, 0, &states[tid]);
  out_I[tid] = hiprand_uniform(&states[tid])*96;
}

__global__ void random_I_unique(uint8_t *out_I,uint8_t *out_k,hiprandState *states){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(clock64(), tid, 0, &states[tid]);
	uint64_t start = 0;
	uint8_t el;
	for(uint64_t i=0;i<tid;i++)
		start+=out_k[i];
	for(uint64_t i=0;i<out_k[tid];i++){
		el = hiprand_uniform(&states[tid])*96;
		while(unsigned8_element_in_array_gpu(el,&out_I[start],out_k[tid])){
			el = hiprand_uniform(&states[tid])*96;
		}
		out_I[start+i] = el;
	}	
}



  


__global__ void generate_IV(uint8_t *cube,uint8_t *I_iv,uint8_t len){
  uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  // each tid map I_i
  IV_gen(tid,len,cube,&I_iv[tid*IV_dim]); // uint64_t tid, uint8_t len, uint8_t cube_index[],uint8_t *iv
}




__global__ void generate_key_set(uint8_t *k,hiprandState *states){
    uint64_t tid = (blockIdx.x * blockDim.x + threadIdx.x);
	hiprand_init(clock64(), tid, 0, &states[tid]);
    k[tid] = hiprand_uniform(&states[tid])*0xFF;
}

__global__ void generate_key_set_xor(uint8_t *k1_set, uint8_t *k2_set, uint8_t *k_xor_set){
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    k_xor_set[tid] = k1_set[tid]^k2_set[tid];
}


__global__ void cuda_encrypt_2_exploit(uint8_t *cube,uint8_t *out,uint64_t window){
	uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint8_t IV[IV_dim],pr=0;
	for(uint64_t i=0;i<window;i++){
		IV_gen(((tid)*window)+i,k_curr_dev,cube,IV);
		pr ^= encrypt_exploit(IV);
	}
	out[tid] = pr;
}



/// Interleaved Pair Implementation with less divergence
__global__ void sumZ2CubeReduceInterleaved (uint8_t *g_idata, uint8_t *g_odata, unsigned int n) {
	// set thread ID
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<n){
	unsigned int tid = threadIdx.x;
	// convert global data pointer to the local pointer of this block 
	uint8_t *idata = g_idata + blockIdx.x * blockDim.x;
	// boundary check if(idx >= n) return;
	// in-place reduction in global memory
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (tid < stride) {
			idata[tid] ^= idata[tid + stride];
		}
		__syncthreads(); 
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = idata[0]; 
	}
}

__global__ void sumCubeReduceInterleaved (uint8_t *g_idata, uint8_t *g_odata, unsigned int n) {
	// set thread ID
	unsigned int id =  blockIdx.x * blockDim.x + threadIdx.x;
	if (id<n){
	unsigned int tid = threadIdx.x;
	//unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// convert global data pointer to the local pointer of this block 
	uint8_t *idata = g_idata + blockIdx.x * blockDim.x;
	// boundary check if(idx >= n) return;
	// in-place reduction in global memory
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (tid < stride) {
			idata[tid] += idata[tid + stride];
		}
		__syncthreads(); 
	}
	// write result for this block to global mem
	if (tid == 0) g_odata[blockIdx.x] = idata[0]; 
	}
}


__global__ void cudaTestBLR(uint8_t *out,uint8_t* p1_set, uint8_t *p2_set, uint8_t *p1_2_set, unsigned int n){
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid<n){
		out[tid] = (uint8_t)((p1_set[tid]^p2_set[tid] != p1_2_set[tid]) && (d_p0_dev^p1_set[tid]^p2_set[tid]!=p1_2_set[tid]));
		//out[tid] = ((p1_set[tid]^p2_set[tid] != p1_2_set[tid]));

	}
}

void flush_arr(uint8_t *arr, uint64_t len){
	for(uint64_t i=0; i<len; i++)
		arr[i]=0;
}



void print_arr_host(const char *s, uint8_t *k, unsigned int dim){
  printf("\n%s = [",s);
  for(int i=0; i<dim;i++){
	printf(" %u, ",k[i]);
  }
  printf("]\n");
}





void print_arr_host_u64(const char *s, uint64_t *k, unsigned int dim){
    printf("\n%s = [",s);
    for(int i=0; i<dim;i++){
      printf(" %lu, ",k[i]);
    }
    printf("]\n");
  }

void print_arr_IV_host(const char *s, uint8_t *k, unsigned int dim){
	for(int i=0; i<dim;i++){
	  if(i%12==0){
		  if(i!=0) 
		    printf(" ]");
		  printf("\n%s = [",s);
	  }
	  printf(" %u, ",k[i]);
	}
	printf("]\n");
  }

uint64_t sum_k(uint8_t *k, unsigned int dim){
	uint64_t sum=0;
	for(int i=0;i<dim;i++)
		sum+=k[i];
	return sum;
}

uint64_t get_nbytes_IV(uint8_t *k, unsigned int dim){
	uint64_t sum = 0;
	for(int i=0;i<dim;i++)
		sum+=1ull<<k[i];
	return sum;
} 

void copy_data_I(uint8_t *src, uint8_t *dst, unsigned int dim){
	for(int i=0;i<dim;i++)
		dst[i] = src[i];
}

void random_key(uint8_t k[]){
	for(int i=0;i<K_dim;i++)
		k[i]=rand()%(1<<8);
}

void xor_key_host(uint8_t k_0[],uint8_t k_1[],uint8_t k_xor[]){
	for(int i=0;i<K_dim;i++)
		k_xor[i] = k_0[i]^k_1[i];
}


void print_IV_host(uint8_t iv[]){
	printf("\n IV vector = [ ");
	for(int i=0;i<IV_dim;i++){
		printf("%u ",iv[i]);
	}
	printf("]\n");
}

void print_key_host(uint8_t k[]){
	printf("\n K vector = [ ");
	for(int i=0;i<K_dim;i++){
		printf("%u ",k[i]);
	}
	printf("]\n");
}

void copy_IV_host(uint8_t* dst, uint8_t* src){
	for(int i=0; i<IV_dim; i++)
		dst[i]=src[i];
}

bool equal_IV(uint8_t *iv,uint8_t *iv1){
	bool r=true;
	for(int i=0; i<IV_dim; i++){
		if(iv[i]!=iv1[i]){
			r = false;
			break;
		}
	}
	return r; 
}


bool signed64_element_in_array(int64_t el, int64_t *arr, uint64_t len){
	bool found = false;
	for(int i=0 ;i<len;i++){
		if(arr[i] == el){
			found = true;
			break;
		}
	}
	return found;
}

bool unsigned64_element_in_array(uint64_t el, uint64_t *arr, uint64_t len){
	bool found = false;
	for(int i=0 ;i<len;i++){
		if(arr[i] == el){
			found = true;
			break;
		}
	}
	return found;
}


bool unsigned8_element_in_array(uint8_t el, uint8_t *arr, uint64_t len){
	bool found = false;
	for(int i=0 ;i<len;i++){
		if(arr[i] == el){
			found = true;
			break;
		}
	}
	return found;
}

uint64_t get_IV_line(uint8_t *iv, uint8_t *h_IV_host,uint64_t dim,int64_t *excluded,int64_t c){
	int64_t r = -1;
	for(uint64_t i=0;i<dim;i++){
		if(!signed64_element_in_array(i,excluded,c) && equal_IV(iv, &h_IV_host[i*IV_dim])){
			r = i;
			break;
		}
	}
	return r;
}



// h_IV,h_IV_host,n_bytes_IV*IV_dim
bool check_kernel3_results(uint8_t *h_IV, uint8_t *h_IV_host, uint64_t dim){
	int64_t j,c=0;
	bool r = true;
	int64_t *excluded = (int64_t *)malloc(dim*sizeof(int64_t));
	for(int i=0;i<dim;i++)
		excluded[i]=-1;
	for(int i=0;i<dim;i++){
		j = get_IV_line(&h_IV[i*IV_dim],h_IV_host,dim,excluded,c);
		if(j == -1){
			printf("\nNOT FOUND\n");
			print_IV_host(&h_IV[i*IV_dim]);
			r = false;
			break;
		}
		excluded[c] = j;
		c+=1;
	}
	return r;

}

uint8_t *set_cubes(uint8_t *out_I_host,uint8_t *out_k_host,uint64_t n_bytes,uint64_t sum, uint64_t ACTIVE_INSTANCES){
	uint64_t start = 0, start_I = 0;
	uint8_t *h_IV,*h_I,*dev_IV,*dev_I;
	uint64_t data_len = sum,blocksize = 32;


	dim3 block;
	dim3 grid;
	

	// creazione degli stream asincroni non-NULL


	// host alloc and cuda malloc in one time
	CHECK(hipHostAlloc((void**) &h_IV,n_bytes*(sizeof(uint8_t)*IV_dim),hipHostMallocDefault));
	CHECK(hipHostAlloc((void**) &h_I,sum*(sizeof(uint8_t)),hipHostMallocDefault));
	copy_data_I(out_I_host,h_I,sum);
	// copy from host memory to pinned memory
	CHECK(hipMalloc((void **)&dev_IV,n_bytes*(sizeof(uint8_t)*IV_dim)));
	CHECK(hipMalloc((void **)&dev_I,sum*sizeof(uint8_t)));
	hipStream_t stream[ACTIVE_INSTANCES];
	for (int i = 0; i < ACTIVE_INSTANCES; ++i) 
    	CHECK(hipStreamCreate(&stream[i]));
    
	for(int i=0;i<ACTIVE_INSTANCES;i++){
		/*
			foreach INSTANCES (which means each time you consider a different k choice)
			I allocate a kernel with a number of thread equal to the number of cube rotation
			however you need cube to convert tid to the conversion convention
		*/
		data_len = 1ull<<out_k_host[i]; // have to do this for each I - stream
		// set optimal block size for each CUDA stream
		blocksize = (data_len<blocksize)? data_len:blocksize;
		print_arr_host("I host",&h_I[start_I],out_k_host[i]);
		block.x = blocksize;
		grid.x = (data_len + block.x - 1) / block.x;
		CHECK(hipMemcpyAsync(&dev_I[start_I],&h_I[start_I],out_k_host[i]*sizeof(uint8_t),hipMemcpyHostToDevice,stream[i]));
		generate_IV<<<grid,block,0,stream[i]>>>(&dev_I[start_I],&dev_IV[start*IV_dim],out_k_host[i]);
		CHECK(hipMemcpyAsync(&h_IV[start*IV_dim],&dev_IV[start*IV_dim],data_len*sizeof(uint8_t)*IV_dim,hipMemcpyDeviceToHost,stream[i]));
		start+=data_len;
		start_I+=out_k_host[i];
	}	

	for(int i=0; i<ACTIVE_INSTANCES; ++i)
		hipStreamDestroy(stream[i]);
	
	CHECK(hipHostFree(h_I));
	CHECK(hipFree(dev_I));
	CHECK(hipFree(dev_IV));

	
	return h_IV;
}


// set_cubes_host(out_I_host,out_k_host,n_bytes_IV,k_sum);

// uint8_t * as return type

uint8_t *set_cubes_host(uint8_t* out_I_host,uint8_t* out_k_host, uint64_t n_bytes, uint64_t ksum,uint64_t ACTIVE_INSTANCES){
	
	uint8_t *cube_index, *h_IV;
	uint8_t iv[IV_dim],c_i;
	uint64_t len,l_limit,start=0,start_IV=0;

	h_IV = (uint8_t *)malloc(n_bytes*(sizeof(uint8_t)*IV_dim));

	for (int c = 0; c<ACTIVE_INSTANCES; c++){
		len = out_k_host[c]; // I size
		cube_index = &out_I_host[start];
		flush_iv_host(iv);
		l_limit=len;
		l_limit = (1ull<<(l_limit));

		for(unsigned long l=0;l<l_limit;l++){
			for(unsigned int i=0; i<len;i++){
				c_i = ((l/(1ull<<i)))%2;
				//iv[(IV_dim-1)-(cube_index[i]/8)] += to_MSB_host(c_i * (1<<(cube_index[i]%8)));
				iv[(IV_dim-1)-(cube_index[i]/8)] += (c_i * (1<<(cube_index[i]%8)));
			}
			copy_IV_host(&h_IV[start_IV],iv);
			flush_iv_host(iv);	
			start_IV += IV_dim*sizeof(uint8_t); 
		}
		start+=len;
	}

	return h_IV;
}




uint64_t update_length_cubes( uint8_t *out_k_host,uint64_t *del1,uint64_t *add1,uint64_t c,uint64_t del_1_counter,uint64_t add_1_counter){
	uint64_t dim=0;
	for(uint64_t i=0;i<c;i++){
		
		// check if we have to add element in cube
		if(unsigned64_element_in_array(i,add1,add_1_counter))
			dim+=out_k_host[i]+1;

		// check if we have to remove element in cube
		if(unsigned64_element_in_array(i,del1,del_1_counter))
			dim+=out_k_host[i]-1;
	}
	return dim;
}

uint8_t random_I_element(uint8_t *excluded,uint64_t dim){
	uint8_t element = rand()%(IV_dim*8);
	while(unsigned8_element_in_array(element,excluded,dim))
		element = rand()%(IV_dim*8);
	return element;
}


void expand_cube(uint8_t *out_I_host_new,uint8_t *out_I_host,uint64_t dim){
	for(int i=0;i<dim;i++)
		out_I_host_new[i]=out_I_host[i];
	out_I_host_new[dim] = random_I_element(out_I_host,dim);
}


void reduce_cube(uint8_t *out_I_host_new,uint8_t *out_I_host,uint64_t dim){
	uint64_t pos = rand()%dim;
	uint8_t t;
	t = out_I_host[dim-1];
	out_I_host[dim-1] = out_I_host[pos];
	out_I_host[pos] = t;
	for(int i=0;i<dim-1;i++)
		out_I_host_new[i]=out_I_host[i];
}

void print_cube(uint8_t *out_I_host,uint64_t dim){
    printf("\n cube [ ");
    for(int index=0;index<dim;index++){
        printf("%u, ",out_I_host[index]);
    }
    printf("]\n");
}

uint8_t **update_out_k_host(uint8_t *out_I_host, uint8_t *out_k_host,uint64_t *del1,uint64_t *add1,uint64_t del_1_counter,uint64_t add_1_counter, uint64_t c){
  
	uint64_t new_cube_d=0,start=0,start_new=0,k_dim_new = update_length_cubes(out_k_host,del1,add1,c,del_1_counter,add_1_counter);
	printf("\nk_dim_new = %ld\n",k_dim_new);
	uint8_t *out_I_host_new = (uint8_t *)malloc(k_dim_new*sizeof(uint8_t)), *out_k_host_new = (uint8_t *)malloc((add_1_counter+del_1_counter)*sizeof(uint8_t));
	uint8_t **res = (uint8_t **)malloc(sizeof(uint8_t *)*2);

	for(uint64_t d=0; d<c; d++){
		
		if(unsigned64_element_in_array(d,add1,add_1_counter)){
			// check if we have to add element in cube
            if(out_k_host[d]<IV_dim*8){
				out_k_host_new[new_cube_d] = out_k_host[d]+1;
            	printf("\nBefore add element in cube\n");
            	print_cube(&out_I_host[start],out_k_host[d]);
				expand_cube(&out_I_host_new[start_new],&out_I_host[start],out_k_host[d]);
            	printf("\nAfter add element in cube\n");
            	print_cube(&out_I_host_new[start_new],out_k_host_new[new_cube_d]);
				start_new+=out_k_host_new[new_cube_d];
				new_cube_d+=1;
			}
			else	
				printf("\nCube to huge to expand\n");
          
		}
		if(unsigned64_element_in_array(d,del1,del_1_counter)){
			// check if we have to remove element in cube
			out_k_host_new[new_cube_d] = out_k_host[d]-1;
            printf("\nBefore remove element in cube\n");
            print_cube(&out_I_host[start],out_k_host[d]);
			reduce_cube(&out_I_host_new[start_new],&out_I_host[start],out_k_host[d]);
            printf("\nAfter remove element in cube\n");
            print_cube(&out_I_host_new[start_new],out_k_host_new[new_cube_d]);
			start_new+=out_k_host_new[new_cube_d];
			new_cube_d+=1;
		}
		//update new out_k_host and out_I_host
		start+=out_k_host[d];
  	}
	free(out_k_host);
	free(out_I_host);
	res[0] = out_k_host_new;
	res[1] = out_I_host_new;
	return res;                                                               
}

uint8_t check_p_coeff_1(uint8_t *IV,uint64_t len){
	uint8_t gpu_sum=0,c;
	for(int i=0;i<len;i++){
		c = encrypt_exploit_host(&IV[i*IV_dim]);
		gpu_sum ^= c;
	}
	return gpu_sum;
}

uint8_t check_p_coeff(uint8_t *h_I,uint64_t len,uint8_t k_len){
	uint8_t gpu_sum=0,c;
	uint8_t IV[IV_dim];
	for(int i=0;i<len;i++){
		IV_gen_host(i,k_len,h_I,IV);
		c = encrypt_exploit_host(IV);
		gpu_sum ^= c;
	}
	return gpu_sum;
}


bool hostTestBLR(uint8_t gpu_sum_0,uint8_t *p1_set,uint8_t *p2_set,uint8_t *p1_2_set){
	bool test=false;
	for(int i=0;i<3*N;i++){
		if(p1_set[i]^p2_set[i] != p1_2_set[i] && gpu_sum_0^p1_set[i]^p2_set[i]!=p1_2_set[i]){
			test=true;
			break;
		}
	}
	return test;
}

uint64_t count_response(uint8_t gpu_sum_0,uint8_t *p1_set,uint8_t *p2_set){
	uint64_t count=0;
	for(int i=0;i<3*N;i++){
		if(p1_set[i]==1) count+=1;
		if(p2_set[i]==1) count+=1;
	}
	count+=gpu_sum_0;
	return count;
}

uint64_t len_cube(char *s, size_t len){
	uint64_t count = 0;
	for(int i = 0; i<len; i++){
		if(s[i]==',')
			count++;
	}
	count++;
	return count;
}

uint8_t* get_cube(char *s, size_t len){
	uint64_t lcs = len_cube(s,len);
	int temp;
	uint8_t *c = (uint8_t *)malloc(lcs*sizeof(uint8_t));
	char buf[4],b_i=0,c_i=0;
	for(int i=0;i<len+1;i++){
		if(s[i]!=',' && s[i]!='\n' && s[i]!='\0' && s[i]!=EOF && i!=len){
			buf[b_i]=s[i];
			b_i++;
		}
		else{
			buf[b_i]='\0';
			if(strlen(buf)!=0){
			temp = atoi(buf);
			c[c_i] = (uint8_t)temp;
			b_i=0;
			c_i++;}
		}
	
	}

	return c;

}


uint8_t* get_cube_uint8(uint8_t *s, size_t len){
	uint64_t lcs = K_dim;
	int temp;
	uint8_t *c = (uint8_t *)malloc(lcs*sizeof(uint8_t));
	char buf[4],b_i=0,c_i=0;
	for(int i=0;i<len+1;i++){
		if(s[i]!=',' && s[i]!='\n' && s[i]!='\0' && s[i]!=(uint64_t)EOF && i!=len){
			buf[b_i]=s[i];
			b_i++;
		}
		else{
			buf[b_i]='\0';
			if(strlen(buf)!=0){
			temp = atoi(buf);
			c[c_i] = (uint8_t)temp;
			b_i=0;
			c_i++;}
		}
	
	}

	return c;

}

uint8_t *merge_cube(uint8_t *cubes,uint8_t *new_cube,uint64_t len_cubes,uint64_t len_new_cube){
	cubes = (uint8_t *)realloc(cubes,(len_new_cube+len_cubes)*sizeof(uint8_t));
	for(uint64_t i=len_cubes;i<len_cubes+len_new_cube;i++){
		cubes[i] = new_cube[i-len_cubes];
	}
	return cubes;

	
}

void print_cubes(cubes c){

	printf("\n######## CUBES ########\n");
	uint64_t base=0;

	for(int i=0;i<c.n_cubes;i++){
		for(int i1=0;i1<c.cubes_len[i];i1++){
			printf("%u ",c.cubes[base+i1]);
		}
		base+=c.cubes_len[i];
		printf("\n");
	}
	printf("\n#######################\n");

}

//h_key_2_guess
void get_key(){

	char fname[] = "./final_attack/offline/key2guess.txt";
	FILE *fp = fopen(fname,"r");
	if(fp == NULL){
		printf("\nError fopen in get key\n");
		exit(EXIT_FAILURE);
		
	}
	char * line = NULL;
	ssize_t read;
	size_t len=0;
	uint8_t *hk;
	uint64_t len_line;
	if((read = getline(&line, &len, fp)) != -1){
		len_line = len_cube(line,read);
		hk= get_cube(line,read);
	}
	else
		printf("\nError in readline of secret key\n");

	for(int i=0;i<K_dim;i++){
		h_key_2_guess[i]=hk[i];
	}
}

void get_cubes(cubes *c){
	c->cubes = NULL;
	c->cubes_len = NULL;
	char fname[] = "./final_attack/offline/cubes_test_val.txt";
	int ln = 0;
	FILE *fp = fopen(fname,"r");
	
	char * line = NULL;
	
	size_t len=0;
	ssize_t read;
	uint64_t len_cubes=0,len_new_cube=0,n_cubes=0;
	if(fp == NULL){
		printf("\nError fopen in get cubes\n");
		exit(EXIT_FAILURE);
		
	}
	while((read = getline(&line, &len, fp)) != -1){
		len_new_cube = len_cube(line, read);
		c->cubes=merge_cube(c->cubes,get_cube(line, read),len_cubes,len_new_cube);
		len_cubes+=len_new_cube;
		n_cubes++;
		c->cubes_len=(uint8_t *)realloc(c->cubes_len,n_cubes*sizeof(uint8_t));
		c->cubes_len[n_cubes-1] = len_new_cube;
		ln++;

	}
	fclose(fp);
	if(line) free(line);
	c->n_cubes = n_cubes;
}

void fprint_right_sp(FILE *fp, uint8_t *out_I_host,uint64_t dim){
    for(int index=0;index<dim;index++){
        fprintf(fp,"%u%s",out_I_host[index],"\n");
    }

}

int main(int argc, char *argv[]){

	/*measure time*/
	
	get_key();
	print_key_host(h_key_2_guess);

	cubes cs;
	get_cubes(&cs);
	print_cubes(cs);
	printf("\nOK\n");

	uint8_t *dev_reduce_out_n,*hreduce_out_n=NULL,*dev_out_cube_n;
	//uint8_t k0[K_dim*sizeof(uint8_t)]={0x0},gpu_sum_0;
	uint8_t gpu_sum_0;
	uint64_t start=0,data_len=0,start_I=0;
	unsigned int blocksize = 32;
	dim3 block(blocksize, 1);
	dim3 grid((INSTANCES + block.x - 1) / block.x, 1);

	uint8_t *dev_I;

	uint64_t k_sum = sum_k(cs.cubes_len,cs.n_cubes);
	uint64_t n_bytes_IV = get_nbytes_IV(cs.cubes_len,cs.n_cubes);
	printf("\nk_sum = %lu,n bytes = %lu\n",k_sum,n_bytes_IV);
	print_arr_host("out_k ",cs.cubes_len,cs.n_cubes);
	uint8_t *h_IV = set_cubes(cs.cubes,cs.cubes_len,n_bytes_IV,k_sum,cs.n_cubes);

	
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_key_2_guess), &h_key_2_guess, K_dim*sizeof(uint8_t)));


	//CHECK(hipMalloc((void **)&out_cube,n_bytes_IV*(sizeof(uint8_t))));
	//CHECK(hipMalloc((void **)&reduce_out,n_bytes_IV*(sizeof(uint8_t))));


	CHECK(hipMalloc((void **)&dev_I,k_sum*sizeof(uint8_t)));
	
	uint8_t right_superpoly[cs.n_cubes];
	uint64_t window;

	for(int c=0;c<cs.n_cubes;c++){

		data_len = (uint64_t)1ull<<(uint64_t)cs.cubes_len[c];
		/*blocksize=32;
		blocksize = (data_len<blocksize)? data_len:blocksize;*/
		if(data_len/(1ull<<24) > 0) // 2*25
			blocksize = 1024;
		else	
			blocksize=32;

		window = ((data_len/(1ull<<30))>0)? data_len/(1ull<<30):1;
		data_len/=window;
		blocksize = (data_len<blocksize)? data_len:blocksize;


		block.x = blocksize;
		grid.x = (data_len+ block.x - 1) / block.x;

		hreduce_out_n =(uint8_t *)realloc(hreduce_out_n,grid.x*sizeof(uint8_t));
		hipMemcpyToSymbol(*(&k_curr_dev),&cs.cubes_len[c], sizeof(uint8_t));
		printf("\ncubes len = %u, start_I = %lu\n",cs.cubes_len[c],start_I);
		print_arr_host("K_host",&cs.cubes[start_I],cs.cubes_len[c]);
		CHECK(hipMemcpyAsync(&dev_I[start_I],&cs.cubes[start_I],cs.cubes_len[c]*sizeof(uint8_t),hipMemcpyHostToDevice));
		CHECK(hipDeviceSynchronize());

		// compute for k0
		CHECK(hipMalloc((void **)&dev_out_cube_n,data_len*sizeof(uint8_t)));
		CHECK(hipMalloc((void **)&dev_reduce_out_n,data_len*sizeof(uint8_t)));
		//CHECK(hipMemcpy((void **)&d_k,(void **)&k0, sizeof(uint8_t)*K_dim, hipMemcpyHostToDevice));

		cuda_encrypt_2_exploit<<<grid,block>>>(&dev_I[start_I],dev_out_cube_n,window);
		CHECK(hipDeviceSynchronize());
		sumZ2CubeReduceInterleaved<<<grid,block>>>(dev_out_cube_n,dev_reduce_out_n,data_len);
		CHECK(hipDeviceSynchronize());
		
		for(int i=0;i<grid.x;i++)
			hreduce_out_n[i]=0;
		
		CHECK(hipMemcpy(hreduce_out_n, dev_reduce_out_n, grid.x  * sizeof(uint8_t),hipMemcpyDeviceToHost));

		
		
		gpu_sum_0 = 0;
		for (int i = 0; i < grid.x; i++)
			gpu_sum_0 ^= hreduce_out_n[i];
			 
		/*
		if(!check_p_coeff(&cs.cubes[start_I],data_len,cs.cubes_len[c])==gpu_sum_0){
			printf("\n[FAIL] secret k fail\n");
			exit(1);
		}*/
		
		printf("\n[%d] gpu sum => %u\n",c,gpu_sum_0);
		
		right_superpoly[c] = gpu_sum_0;
		CHECK(hipFree(dev_out_cube_n));
		CHECK(hipFree(dev_reduce_out_n));

		start+=data_len;
		start_I+=cs.cubes_len[c];
		//start_IV+=(data_len*IV_dim);
	}

	printf("\n############################\n");

	for(int i=0;i<cs.n_cubes;i++){
		printf("\n[%d] gpu sum => %u\n",i,right_superpoly[i]);

	} 
	printf("\n############################\n");

	/*
    FILE *fp = fopen("./final_attack/offline/superpolies_right_val.txt","w");

    fprint_right_sp(fp,right_superpoly,cs.n_cubes);
    */

	CHECK(hipDeviceReset());

	return 0;

}

